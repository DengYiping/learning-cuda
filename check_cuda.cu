#include <stdio.h>
#include "hip/hip_runtime.h"

int main() {
    int device_id = 0; // Or the ID you expect the H100 to be
    hipDeviceProp_t properties;
    hipError_t prop_err = hipGetDeviceProperties(&properties, device_id);
    if (prop_err != hipSuccess) {
         fprintf(stderr, "Failed to get properties for device %d: %s\n", device_id, hipGetErrorString(prop_err));
         return 1;
    }
    printf("Device %d Name: %s\n", device_id, properties.name);
    printf("Compute Capability: %d.%d\n", properties.major, properties.minor);


    int available_shared_memory = 0;
    hipError_t attr_err = hipDeviceGetAttribute(&available_shared_memory, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id);
     if (attr_err != hipSuccess) {
         fprintf(stderr, "Failed to get attribute for device %d: %s\n", device_id, hipGetErrorString(attr_err));
         return 1;
    }

    printf("hipDeviceAttributeMaxSharedMemoryPerBlock for device %d: %d bytes (%d KiB)\n",
           device_id, available_shared_memory, available_shared_memory / 1024);

    return 0;
}