#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#define M 8192
#define N 4096
#define K 2048

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}

#undef PRINT_MATRIX
#define PRINT_MATRIX(mat, rows, cols) \
    printf("Top-left 3x3 corner:\n"); \
    for (int i = 0; i < 3 && i < rows; i++) { \
        for (int j = 0; j < 3 && j < cols; j++) \
            printf("%8.3f ", mat[i * cols + j]); \
        printf("\n"); \
    } \
    printf("\n");

void cpu_matmul(float *A, float *B, float *C) {
    // For large matrices, only compute the top-left 3x3 corner for verification
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++)
                sum += A[i * K + k] * B[k * N + j];
            C[i * 3 + j] = sum;
        }
    }
}

int main(int argc, char** argv) {
  srand(time(NULL));
  
  // Allocate host memory for large matrices using pinned memory
  float *A, *B, *h_cpu, *h_cublaslt_s, *h_cublaslt_h, *h_cublaslt_bf16;
  
  CHECK_CUDA(hipHostMalloc(&A, M * K * sizeof(float), hipHostMallocDefault));
  CHECK_CUDA(hipHostMalloc(&B, K * N * sizeof(float), hipHostMallocDefault));
  CHECK_CUDA(hipHostMalloc(&h_cpu, 9 * sizeof(float), hipHostMallocDefault));  // Only need 3x3 for CPU verification
  CHECK_CUDA(hipHostMalloc(&h_cublaslt_s, M * N * sizeof(float), hipHostMallocDefault));
  CHECK_CUDA(hipHostMalloc(&h_cublaslt_h, M * N * sizeof(float), hipHostMallocDefault));
  CHECK_CUDA(hipHostMalloc(&h_cublaslt_bf16, M * N * sizeof(float), hipHostMallocDefault));

  if (!A || !B || !h_cpu || !h_cublaslt_s || !h_cublaslt_h || !h_cublaslt_bf16) {
    fprintf(stderr, "Memory allocation failed\n");
    exit(EXIT_FAILURE);
  }

  // Initialize matrices with random values
  for (int i = 0; i < M * K; i++) {
    A[i] = (float)rand() / RAND_MAX;
  }
  for (int i = 0; i < K * N; i++) {
    B[i] = (float)rand() / RAND_MAX;
  }

  // CPU matmul for verification (only computes top-left 3x3)
  cpu_matmul(A, B, h_cpu);

  // cublasLt setup
  hipblasLtHandle_t ltHandle;
  CHECK_CUBLAS(hipblasLtCreate(&ltHandle));

  // Mem setup
  float *d_A, *d_B, *d_C;
  CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));
  CHECK_CUDA(hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice));

  // cublasLt SGEMM
  float alpha = 1.0f, beta = 0.0f;

  // Create matrix descriptors for B^T, A^T, C^T (treating row-major input as col-major transpose)
  // B (KxN row-major) -> B^T (NxK col-major), ld = N
  // A (MxK row-major) -> A^T (KxM col-major), ld = K
  // C (MxN row-major) -> C^T (NxM col-major), ld = N
  hipblasLtMatrixLayout_t matB_T, matA_T, matC_T;
  CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_T, HIP_R_32F, N, K, N)); // B^T is NxK, leading dim N
  CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_T, HIP_R_32F, K, M, K)); // A^T is KxM, leading dim K
  CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_T, HIP_R_32F, N, M, N)); // C^T is NxM, leading dim N

  // Create operation descriptor
  hipblasLtMatmulDesc_t operationDesc;
  CHECK_CUBLAS(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
  hipblasOperation_t opN = HIPBLAS_OP_N;
  CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opN, sizeof(opN)));
  CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opN, sizeof(opN)));

  // Create preference handle
  hipblasLtMatmulPreference_t preference;
  CHECK_CUBLAS(hipblasLtMatmulPreferenceCreate(&preference));
  size_t workspaceSize = 4 * 1024 * 1024;  // 4 MB, use size_t
  CHECK_CUBLAS(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

  // Find the best algorithm
  int returnedAlgoCount = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  CHECK_CUBLAS(hipblasLtMatmulAlgoGetHeuristic(
      ltHandle, operationDesc, matB_T, matA_T, matC_T, matC_T, preference, 1, &heuristicResult, &returnedAlgoCount));

  if (returnedAlgoCount == 0) {
    fprintf(stderr, "No algorithm returned\n");
    exit(EXIT_FAILURE);
  }

  // Execute cublasLt GEMM for FP32: Computes C^T = B^T * A^T
  CHECK_CUBLAS(hipblasLtMatmul(
      ltHandle, operationDesc, &alpha,
      d_B, matB_T,  // B^T (data from d_B)
      d_A, matA_T,  // A^T (data from d_A)
      &beta,
      d_C, matC_T,  // C^T (output)
      d_C, matC_T,  // C^T (output)
      &heuristicResult.algo, NULL, 0, 0));

  CHECK_CUDA(hipMemcpy(h_cublaslt_s, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost)); // M*N is correct size for C

  // For FP16 - create new matrices and descriptors
  half *h_A_h, *h_B_h;
  CHECK_CUDA(hipHostMalloc(&h_A_h, M * K * sizeof(half), hipHostMallocDefault));
  CHECK_CUDA(hipHostMalloc(&h_B_h, K * N * sizeof(half), hipHostMallocDefault));
  
  if (!h_A_h || !h_B_h) {
    fprintf(stderr, "Memory allocation for half precision input arrays failed\n");
    exit(EXIT_FAILURE);
  }
  
  half *d_A_h, *d_B_h;
  float *d_C_h;
  CHECK_CUDA(hipMalloc(&d_A_h, M * K * sizeof(half)));
  CHECK_CUDA(hipMalloc(&d_B_h, K * N * sizeof(half)));
  CHECK_CUDA(hipMalloc(&d_C_h, M * N * sizeof(float)));

  // Convert float to half
  for (int i = 0; i < M * K; i++) {
    h_A_h[i] = __float2half(A[i]);
  }
  for (int i = 0; i < K * N; i++) {
    h_B_h[i] = __float2half(B[i]);
  }
  CHECK_CUDA(hipMemcpy(d_A_h, h_A_h, M * K * sizeof(half), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_B_h, h_B_h, K * N * sizeof(half), hipMemcpyHostToDevice));

  // FP16 descriptors for B^T (FP16), A^T (FP16), C^T (FP32)
  hipblasLtMatrixLayout_t matB_T_h, matA_T_h, matC_T_h;
  CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_T_h, HIP_R_16F, N, K, N)); // B^T is NxK (FP16), ld N
  CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_T_h, HIP_R_16F, K, M, K)); // A^T is KxM (FP16), ld K
  CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_T_h, HIP_R_32F, N, M, N)); // C^T is NxM (FP32), ld N

  // Create operation descriptor for FP16 inputs, FP32 compute/output/scale
  hipblasLtMatmulDesc_t operationDesc_h;
  CHECK_CUBLAS(hipblasLtMatmulDescCreate(&operationDesc_h, HIPBLAS_COMPUTE_32F, HIP_R_32F)); // <-- Changed scale type to FP32
  CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc_h, HIPBLASLT_MATMUL_DESC_TRANSA, &opN, sizeof(opN)));
  CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc_h, HIPBLASLT_MATMUL_DESC_TRANSB, &opN, sizeof(opN)));

  // Find the best algorithm for FP16: B^T * A^T -> C^T
  hipblasLtMatmulHeuristicResult_t heuristicResult_h = {};
  CHECK_CUBLAS(hipblasLtMatmulAlgoGetHeuristic(
      ltHandle, operationDesc_h, matB_T_h, matA_T_h, matC_T_h, matC_T_h, preference, 1, &heuristicResult_h, &returnedAlgoCount));

  if (returnedAlgoCount == 0) {
    fprintf(stderr, "No algorithm returned for FP16\n");
    exit(EXIT_FAILURE);
  }

  // Execute cublasLt GEMM for FP16 inputs, FP32 output: Computes C^T = B^T * A^T
  CHECK_CUBLAS(hipblasLtMatmul(
      ltHandle, operationDesc_h, &alpha,
      d_B_h, matB_T_h, // B^T (data from d_B_h, FP16)
      d_A_h, matA_T_h, // A^T (data from d_A_h, FP16)
      &beta,          // Use float beta
      d_C_h, matC_T_h, // C^T (output, FP32)
      d_C_h, matC_T_h, // C^T (output, FP32)
      &heuristicResult_h.algo, NULL, 0, 0));

  // Copy results back (already float)
  CHECK_CUDA(hipMemcpy(h_cublaslt_h, d_C_h, M * N * sizeof(float), hipMemcpyDeviceToHost)); // Copy directly to float host buffer

  // For BF16 - create new matrices and descriptors
  __hip_bfloat16 *h_A_bf16, *h_B_bf16;
  CHECK_CUDA(hipHostMalloc(&h_A_bf16, M * K * sizeof(__hip_bfloat16), hipHostMallocDefault));
  CHECK_CUDA(hipHostMalloc(&h_B_bf16, K * N * sizeof(__hip_bfloat16), hipHostMallocDefault));
  
  if (!h_A_bf16 || !h_B_bf16) {
    fprintf(stderr, "Memory allocation for bfloat16 arrays failed\n");
    exit(EXIT_FAILURE);
  }
  
  __hip_bfloat16 *d_A_bf16, *d_B_bf16;
  float *d_C_bf16; // Using float for output
  CHECK_CUDA(hipMalloc(&d_A_bf16, M * K * sizeof(__hip_bfloat16)));
  CHECK_CUDA(hipMalloc(&d_B_bf16, K * N * sizeof(__hip_bfloat16)));
  CHECK_CUDA(hipMalloc(&d_C_bf16, M * N * sizeof(float)));

  // Convert float to bfloat16
  for (int i = 0; i < M * K; i++) {
    h_A_bf16[i] = __float2bfloat16(A[i]);
  }
  for (int i = 0; i < K * N; i++) {
    h_B_bf16[i] = __float2bfloat16(B[i]);
  }
  CHECK_CUDA(hipMemcpy(d_A_bf16, h_A_bf16, M * K * sizeof(__hip_bfloat16), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_B_bf16, h_B_bf16, K * N * sizeof(__hip_bfloat16), hipMemcpyHostToDevice));

  // BF16 descriptors for B^T, A^T, C^T
  // Inputs B^T (BF16), A^T (BF16), Output C^T (F32)
  hipblasLtMatrixLayout_t matB_T_bf16, matA_T_bf16, matC_T_f32;
  CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_T_bf16, HIP_R_16BF, N, K, N)); // B^T (BF16) NxK, ld N
  CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_T_bf16, HIP_R_16BF, K, M, K)); // A^T (BF16) KxM, ld K
  CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_T_f32, HIP_R_32F, N, M, N)); // C^T (F32) NxM, ld N

  // Create operation descriptor for BF16 inputs, F32 compute/output
  hipblasLtMatmulDesc_t operationDesc_bf16;
  CHECK_CUBLAS(hipblasLtMatmulDescCreate(&operationDesc_bf16, HIPBLAS_COMPUTE_32F, HIP_R_32F)); // Compute F32, Scale F32
  CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc_bf16, HIPBLASLT_MATMUL_DESC_TRANSA, &opN, sizeof(opN)));
  CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc_bf16, HIPBLASLT_MATMUL_DESC_TRANSB, &opN, sizeof(opN)));

  // Find the best algorithm for BF16: B^T * A^T -> C^T
  hipblasLtMatmulHeuristicResult_t heuristicResult_bf16 = {};
  CHECK_CUBLAS(hipblasLtMatmulAlgoGetHeuristic(
      ltHandle, operationDesc_bf16, matB_T_bf16, matA_T_bf16, matC_T_f32, matC_T_f32, preference, 1, &heuristicResult_bf16, &returnedAlgoCount));

  if (returnedAlgoCount == 0) {
    fprintf(stderr, "No algorithm returned for BF16\n");
    exit(EXIT_FAILURE);
  }

  // Execute cublasLt GEMM for BF16 inputs, F32 output: Computes C^T = B^T * A^T
  CHECK_CUBLAS(hipblasLtMatmul(
      ltHandle, operationDesc_bf16, &alpha, // Use float alpha/beta
      d_B_bf16, matB_T_bf16, // B^T (data from d_B_bf16)
      d_A_bf16, matA_T_bf16, // A^T (data from d_A_bf16)
      &beta,                 // Use float alpha/beta
      d_C_bf16, matC_T_f32,  // C^T (output, F32)
      d_C_bf16, matC_T_f32,  // C^T (output, F32)
      &heuristicResult_bf16.algo, NULL, 0, 0));

  CHECK_CUDA(hipMemcpy(h_cublaslt_bf16, d_C_bf16, M * N * sizeof(float), hipMemcpyDeviceToHost)); // M*N is correct size for C

  // Print the results
  printf("CPU result (top-left 3x3 only):\n");
  PRINT_MATRIX(h_cpu, 3, 3);
  printf("cublasLt SGEMM result:\n");
  PRINT_MATRIX(h_cublaslt_s, M, N);
  printf("cublasLt HGEMM result:\n");
  PRINT_MATRIX(h_cublaslt_h, M, N);
  printf("cublasLt GEMM with BF16 result:\n");
  PRINT_MATRIX(h_cublaslt_bf16, M, N);

  // Free resources
  CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matB_T));
  CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matA_T));
  CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matC_T));
  CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matB_T_h));
  CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matA_T_h));
  CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matC_T_h));
  CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matB_T_bf16));
  CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matA_T_bf16));
  CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matC_T_f32));
  CHECK_CUBLAS(hipblasLtMatmulDescDestroy(operationDesc));
  CHECK_CUBLAS(hipblasLtMatmulDescDestroy(operationDesc_h));
  CHECK_CUBLAS(hipblasLtMatmulDescDestroy(operationDesc_bf16));
  CHECK_CUBLAS(hipblasLtMatmulPreferenceDestroy(preference));
  CHECK_CUBLAS(hipblasLtDestroy(ltHandle));

  // Free memory
  CHECK_CUDA(hipHostFree(A));
  CHECK_CUDA(hipHostFree(B));
  CHECK_CUDA(hipHostFree(h_cpu));
  CHECK_CUDA(hipHostFree(h_cublaslt_s));
  CHECK_CUDA(hipHostFree(h_cublaslt_h));
  CHECK_CUDA(hipHostFree(h_cublaslt_bf16));
  CHECK_CUDA(hipHostFree(h_A_h));
  CHECK_CUDA(hipHostFree(h_B_h));
  CHECK_CUDA(hipHostFree(h_A_bf16));
  CHECK_CUDA(hipHostFree(h_B_bf16));

  CHECK_CUDA(hipFree(d_A));
  CHECK_CUDA(hipFree(d_B));
  CHECK_CUDA(hipFree(d_C));
  CHECK_CUDA(hipFree(d_A_h));
  CHECK_CUDA(hipFree(d_B_h));
  CHECK_CUDA(hipFree(d_C_h));
  CHECK_CUDA(hipFree(d_A_bf16));
  CHECK_CUDA(hipFree(d_B_bf16));
  CHECK_CUDA(hipFree(d_C_bf16));

  return 0;
} 