#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "faster_matmul.cuh"
#include <hip/hip_cooperative_groups.h>
#include "ptx.cuh"
#include <hip/hip_runtime.h> // Include CUDA Driver API header
#include <stdio.h> // For printf debugging

#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))
namespace cg = cooperative_groups;

// Helper macro for CUDA Driver API error checking
#define CHECK_CUDA_DRIVER(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        const char *err_str; \
        hipDrvGetErrorString(err, &err_str); \
        fprintf(stderr, "CUDA Driver error in %s at line %d: %s\n", __FILE__, __LINE__, err_str); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

constexpr uint TM = 4;
constexpr uint TN = 4;
constexpr uint WMITER = 2;
constexpr uint WNITER = 2;

constexpr uint WM = 32;
constexpr uint WN = 64;

constexpr uint W_SUB_M = WM / WMITER; // 16 -> 4 threads
constexpr uint W_SUB_N = WN / WNITER; // 32 -> 8 threads 

template <int BM, int BN, int BK>
__global__ __launch_bounds__(BM * BN / (TM * TN)) void vectorized_2d_block_tiling_matmul(
    const __grid_constant__ CUtensorMap tensor_map_A, // Pass by value with __grid_constant__
    const __grid_constant__ CUtensorMap tensor_map_B, // Pass by value with __grid_constant__
    float* __restrict__ C,
    const int M, const int N, const int K)
{
    // Define aligned shared memory
    alignas(128) extern __shared__ char smem_buffer[]; // Use char for byte-level layout, align to 16 bytes

    // Derive float pointers from the aligned buffer
    float* shared_A = reinterpret_cast<float*>(smem_buffer);
    float* shared_B = shared_A + BM * BK;

    const uint lane_id = threadIdx.x & 0x1F; // 0 ... 31
    const uint warp_id = threadIdx.x >> 5; // 0 ... 31

    const uint warp_col = warp_id % (BN / WN);
    const uint warp_row = warp_id / (BN / WN);

    const uint lane_col = lane_id % (W_SUB_N / TN);
    const uint lane_row = lane_id / (W_SUB_N / TN);

    // Adjust C pointer for the current block
    float* C_block_start = C + blockIdx.y * BM * N + blockIdx.x * BN;

    alignas(16) float thread_results[WMITER * TM][WNITER * TN] = {0.0f};
    alignas(16) float reg_M[TM * WMITER];
    alignas(16) float reg_N[TN * WNITER];

    // PTX Asynchronous copy setup
    auto block = cg::this_thread_block();
    const bool is_master_thread = (block.thread_rank() == 0);
    constexpr int THREADS_PER_BLOCK = BM * BN / (TM * TN);

    alignas(8) __shared__ uint64_t mbar[1];

    if (is_master_thread) {
        ptx::mbarrier_init(&mbar[0], THREADS_PER_BLOCK);
        ptx::fence_mbarrier_init_release_cluster(); // Ensure init is visible to async units
    }
    block.sync(); // Ensure barrier is initialized before use

    uint32_t phase = 0; // Phase for mbarrier wait cycles

    // Outer loop over block tiles in K dimension
    for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
        // Load the next A/B tile from global memory to shared memory using TMA
        constexpr size_t bytes_A_tile = BM * BK * sizeof(float);
        constexpr size_t bytes_B_tile = BK * BN * sizeof(float);
        constexpr size_t total_bytes = bytes_A_tile + bytes_B_tile;

        // Calculate 2D offsets for TMA
        const uint32_t offset_A_x = bkIdx;
        const uint32_t offset_A_y = blockIdx.y * BM;
        const uint32_t offset_B_x = blockIdx.x * BN;
        const uint32_t offset_B_y = bkIdx;

        if (is_master_thread) {
            // Initiate A tile copy (TMA)
            ptx::cp_async_bulk_tensor_2d_global_to_shared(
                reinterpret_cast<uint64_t*>(shared_A),
                (const uint64_t*)&tensor_map_A,
                offset_A_x, offset_A_y,
                &mbar[0]
            );
            // Initiate B tile copy (TMA)
            ptx::cp_async_bulk_tensor_2d_global_to_shared(
                reinterpret_cast<uint64_t*>(shared_B),
                (const uint64_t*)&tensor_map_B,
                offset_B_x, offset_B_y,
                &mbar[0]
            );
            ptx::mbarrier_arrive_expect_tx(&mbar[0], total_bytes);
            if (bkIdx + BK < K) {
                ptx::prefetch_async_bulk_tensor_2d_global_l2(
                    (const uint64_t*)&tensor_map_A,
                    offset_A_x + BK,
                    offset_A_y
                );
                ptx::prefetch_async_bulk_tensor_2d_global_l2(
                    (const uint64_t*)&tensor_map_B,
                    offset_B_x,
                    offset_B_y + BN
                );
            }
        } else {
            ptx::mbarrier_arrive(&mbar[0]);
        }

        // Wait for TMA copies to complete
        ptx::mbarrier_wait_parity(&mbar[0], phase);

        // Perform matrix multiplication for this tile
        #pragma unroll
        for (uint dot_idx = 0; dot_idx < BK; ++dot_idx) {
            // Load column of A and row of B into registers
            for (uint w_sub_row = 0; w_sub_row < WMITER; ++w_sub_row) {
                for (uint i = 0; i < TM; i++) {
                    reg_M[w_sub_row * TM + i] = shared_A[(warp_row * WM + w_sub_row * W_SUB_M + lane_row * TM + i) * BK + dot_idx];
                }
            }

            for (uint w_sub_col = 0; w_sub_col < WNITER; ++w_sub_col) {
                for (uint j = 0; j < TN; j+= 4) {
                    reinterpret_cast<float4*>(reg_N + w_sub_col * TN + j)[0] = 
                        reinterpret_cast<const float4*>(&shared_B[(dot_idx * BN + warp_col * WN + w_sub_col * W_SUB_N + lane_col * TN + j)])[0];
                }
            }

            for (uint w_sub_row = 0; w_sub_row < WMITER; ++w_sub_row) {
                for (uint w_sub_col = 0; w_sub_col < WNITER; ++w_sub_col) {
                    for (uint m = 0; m < TM; ++m) {
                        for (uint n = 0; n < TN; ++n) {
                            thread_results[(w_sub_row * TM + m)][w_sub_col * TN + n] += reg_M[w_sub_row * TM + m] * reg_N[w_sub_col * TN + n];
                        }
                    }
                }
            }
        }

        block.sync(); // Ensure shared mem reads finish before next K-tile TMA overwrite

        phase ^= 1; // Flip phase for next mbarrier wait
    }

    // Store the results
    for (uint w_sub_row = 0; w_sub_row < WMITER; ++w_sub_row) {
        for (uint w_sub_col = 0; w_sub_col < WNITER; ++w_sub_col) {
            for (uint i = 0; i < TM; ++i) {
                for (uint j = 0; j < TN; j+= 4) {
                    uint write_idx_base = 
                        (warp_row * WM + w_sub_row * W_SUB_M + lane_row * TM + i) * N + 
                        (warp_col * WN + w_sub_col * W_SUB_N + lane_col * TN + j);
                    reinterpret_cast<float4*>(&C_block_start[write_idx_base])[0] = reinterpret_cast<float4*>(&thread_results[w_sub_row * TM + i][w_sub_col * TN + j])[0];
                }
            }
        }
    }
}

// Kernel launcher function
void launch_vectorized_2d_block_tiling_matmul(const float* __restrict__ d_A, const float* __restrict__ d_B, float* __restrict__ d_C, int m, int n, int k, hipStream_t stream) {
    constexpr int BM = 64;
    constexpr int BN = 64;
    constexpr int BK = 64;

    // Create Tensor Maps
    CUtensorMap tensor_map_A;
    CUtensorMap tensor_map_B;

    const cuuint32_t elementStrides[] = {1, 1}; // Contiguous access

    // Tensor A (M x K) -> {inner (K), outer (M)}
    const uint64_t globalDimA[] = {(uint64_t)k, (uint64_t)m};
    const uint64_t globalStrideA[] = {sizeof(float), (uint64_t)k * sizeof(float)};
    const cuuint32_t boxDimA[] = {BK, BM};

    CHECK_CUDA_DRIVER(cuTensorMapEncodeTiled(
        &tensor_map_A,
        CU_TENSOR_MAP_DATA_TYPE_FLOAT32,
        2,
        (void*)d_A,
        globalDimA,
        globalStrideA + 1,
        boxDimA,
        elementStrides,
        CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE,
        CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    ));

    // Tensor B (K x N) -> {inner (N), outer (K)}
    const uint64_t globalDimB[] = {(uint64_t)n, (uint64_t)k};
    const uint64_t globalStrideB[] = {sizeof(float), (uint64_t)n * sizeof(float)};
    const cuuint32_t boxDimB[] = {BN, BK};

    CHECK_CUDA_DRIVER(cuTensorMapEncodeTiled(
        &tensor_map_B,
        CU_TENSOR_MAP_DATA_TYPE_FLOAT32,
        2,
        (void*)d_B,
        globalDimB,
        globalStrideB + 1,
        boxDimB,
        elementStrides,
        CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE,
        CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    ));

    // BM * BN / (WM * WN) = number of warps required
    // number of threads = BM * BN * 32 / (WM * WN)
    dim3 blockDim(BM * BN * 32 / (WM * WN));
    dim3 gridDim(CEIL_DIV(n, BN), CEIL_DIV(m, BM));
    size_t shared_mem_size = (BM * BK + BK * BN) * sizeof(float);

    vectorized_2d_block_tiling_matmul<BM, BN, BK><<<gridDim, blockDim, shared_mem_size, stream>>>(
        tensor_map_A,
        tensor_map_B,
        d_C, m, n, k);
}

int main() {
    CHECK_CUDA_DRIVER(hipInit(0));
    hipCtx_t ctx;
    CHECK_CUDA_DRIVER(hipCtxGetCurrent(&ctx));

    constexpr int BM = 64;
    constexpr int BN = 64;
    constexpr int BK = 64;

    hipDeviceProp_t deviceProp;
    CHECK_CUDA(hipGetDeviceProperties(&deviceProp, 0));
    std::cout << "Device name: " << deviceProp.name << std::endl;
    std::cout << "Shared memory size per block: " << deviceProp.sharedMemPerMultiprocessor << std::endl;

    // Set kernel attributes (carveout preference, max dynamic shared memory)
    CHECK_CUDA(hipFuncSetAttribute(
        (const void*)vectorized_2d_block_tiling_matmul<BM, BN, BK>,
        hipFuncAttributePreferredSharedMemoryCarveout,
        90 // 90% of the shared memory per block
    ));
     size_t shared_mem_size = (BM * BK + BK * BN) * sizeof(float);
     if (shared_mem_size > deviceProp.sharedMemPerMultiprocessor) {
         std::cerr << "Warning: Requested shared memory (" << shared_mem_size << ") exceeds device limit (" << deviceProp.sharedMemPerMultiprocessor << ")" << std::endl;
     }
     CHECK_CUDA(hipFuncSetAttribute(
         (const void*)vectorized_2d_block_tiling_matmul<BM, BN, BK>,
         hipFuncAttributeMaxDynamicSharedMemorySize,
         shared_mem_size
     ));

    // Default matrix dimensions
    int m = 4096;
    int n = 2048;
    int k = 512;

    std::cout << "Running Vectorized 2D block tiling (TMA with TensorMap) matrix multiplication benchmark:" << std::endl;

    float avg_time = run_benchmark<float>(
        launch_vectorized_2d_block_tiling_matmul, m, n, k
    );

    return 0;
}