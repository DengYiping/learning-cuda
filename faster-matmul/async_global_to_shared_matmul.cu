#include "hip/hip_runtime.h"
#include "faster_matmul.cuh"
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))

template <int BM, int BN, int BK, int TM, int TN>
__global__ void vectorized_2d_block_tiling_matmul(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, const int M, const int N, const int K) {
    extern __shared__ float shared_A[];
    float* shared_B = shared_A + BM * BK;

    // blockIdx.x is the block id in the N dimension, aka the column index of the block
    // blockIdx.y is the block id in the M dimension, aka the row index of the block

    // Each warp will calculate 32 * TM * TN elements, with 32 being the columnar dim.
    // Num threads = BM * BN / (TM * TN), we will 2d tiling on the M, N dimension.
    const uint thread_col = threadIdx.x % (BN / TN);
    const uint thread_row = threadIdx.x / (BN / TN);

    // Move blocktile to beginning of A's row and B's column
    A += blockIdx.y * BM * K;
    B += blockIdx.x * BN;
    C += blockIdx.y * BM * N + blockIdx.x * BN;

    float thread_results[TM][TN] = {0.0f};
    float reg_M[TM];
    float reg_N[TN];

    // -------------------------  Asynchronous copy helpers -------------------------
    constexpr auto scope = cuda::thread_scope_thread;
    auto thread = cooperative_groups::this_thread();
    auto block  = cooperative_groups::this_thread_block();
    cuda::pipeline<scope> pipe = cuda::make_pipeline();

    // Assume K is divisible by BK. Outer loop is over block tiles
    for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
        /* ------------------------------------------------------------------
           Load the next A/B tile from global memory to shared memory using
           cp.async (cuda::memcpy_async).  We keep exactly one copy of the
           tile in shared memory, therefore shared memory consumption is the
           same as the original kernel.
        ------------------------------------------------------------------ */

        pipe.producer_acquire();

        // load A_tile into shared_A
        for (int idx = threadIdx.x; idx < (BM * BK) / 4; idx += blockDim.x) {
            int row = idx / (BK / 4);       // which row within the tile
            int col4 = idx % (BK / 4);      // which float4 inside the row

            cuda::memcpy_async(
                thread,
                &reinterpret_cast<float4*>(shared_A)[idx],
                &reinterpret_cast<const float4*>(A)[row * (K / 4) + col4],
                cuda::aligned_size_t<16>(sizeof(float4)),
                pipe);
        }

        // load B_tile into shared_B
        for (int idx = threadIdx.x; idx < (BK * BN) / 4; idx += blockDim.x) {
            int row = idx / (BN / 4);
            int col4 = idx % (BN / 4);

            cuda::memcpy_async(
                thread,
                &reinterpret_cast<float4*>(shared_B)[idx],
                &reinterpret_cast<const float4*>(B)[row * (N / 4) + col4],
                cuda::aligned_size_t<16>(sizeof(float4)),
                pipe);
        }

        pipe.producer_commit();

        // Wait until the tile is fully available in shared memory
        pipe.consumer_wait();

        /* ------------------------------------------------------------------
           At this point the entire tile is in shared memory and can be
           consumed by all threads.  The copy from global memory was issued
           asynchronously;  with larger K-tiles the latency will largely be
           hidden behind the arithmetic below while we keep shared memory
           usage unchanged (no double buffering).
        ------------------------------------------------------------------ */

        // Advance global pointers so that they point at the *next* tile in K
        const float* next_A = A + BK;
        const float* next_B = B + BK * N;

        block.sync();
        // Perform matrix multiplication for this tile
        for (uint dot_idx = 0; dot_idx < BK; ++dot_idx) {
            // Load one column of A and one row of B from shared memory into
            // registers and compute a TM x TN outer product.

            for (uint i = 0; i < TM; ++i) {
                reg_M[i] = shared_A[(thread_row * TM + i) * BK + dot_idx];
            }

            for (uint j = 0; j < TN; j += 4) {
                reinterpret_cast<float4*>(&reg_N[j])[0] =
                    reinterpret_cast<float4*>(&shared_B[dot_idx * BN + (thread_col * TN + j)])[0];
            }

            for (uint i = 0; i < TM; ++i) {
                for (uint j = 0; j < TN; ++j) {
                    thread_results[i][j] += reg_M[i] * reg_N[j];
                }
            }
        }

        block.sync();
        // We finished consuming this tile;  let the pipeline know.
        pipe.consumer_release();

        // Move on to the next tile in global memory
        A = next_A;
        B = next_B;
    }

    // Store the results
    for (uint i = 0; i < TM; i++) {
        for (uint j = 0; j < TN; j+= 4) {
            reinterpret_cast<float4*>(&C[(thread_row * TM + i) * N + (thread_col * TN + j)])[0] = reinterpret_cast<float4*>(&thread_results[i][j])[0];
        }
    }
}

// Kernel launcher function
void launch_vectorized_2d_block_tiling_matmul(const float* __restrict__ d_A, const float* __restrict__ d_B, float* __restrict__ d_C, int m, int n, int k, hipStream_t stream) {
    constexpr int BM = 64;
    constexpr int BN = 64;
    constexpr int BK = 64;
    constexpr int TM = 8;
    constexpr int TN = 4;

    // Each thread will calculate TM * TN elements
    dim3 blockDim(BM * BN / (TM * TN)); 
    // Reversing order to optimize L2 cache access. Grid will move on the N dimension fast and M dimension slow.
    // With row-major layout, this is more cache-friendly.
    dim3 gridDim(CEIL_DIV(n, BN), CEIL_DIV(m, BM));
    
    vectorized_2d_block_tiling_matmul<BM, BN, BK, TM, TN><<<gridDim, blockDim, BM * BK * sizeof(float) + BN * BK * sizeof(float), stream>>>(d_A, d_B, d_C, m, n, k);
}

int main() {
    constexpr int BM = 64;
    constexpr int BN = 64;
    constexpr int BK = 64;
    constexpr int TM = 8;
    constexpr int TN = 4;

    hipDeviceProp_t deviceProp;
    CHECK_CUDA(hipGetDeviceProperties(&deviceProp, 0));
    std::cout << "Device name: " << deviceProp.name << std::endl;
    std::cout << "Shared memory size: " << deviceProp.sharedMemPerMultiprocessor << std::endl;
    std::cout << "Shared memory size per block: " << deviceProp.sharedMemPerBlock << std::endl;


    // Set shared memory carveout for this kernel
    CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
        vectorized_2d_block_tiling_matmul<BM), BN, BK, TM, TN>,
        hipFuncAttributePreferredSharedMemoryCarveout,
        75
    ));
    // Set shared memory size for this kernel
    CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
        vectorized_2d_block_tiling_matmul<BM), BN, BK, TM, TN>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        128 * 1024
    ));

    // Default matrix dimensions
    int m = 4096; // Matrix A: m x k
    int n = 2048; // Matrix B: k x n, Matrix C: m x n
    int k = 512;

    std::cout << "Running async_global_to_shared_matmul benchmark:" << std::endl;

    // Run the benchmark with the naive matrix multiplication kernel
    float avg_time = run_benchmark<float>(
        launch_vectorized_2d_block_tiling_matmul, m, n, k
    );

    return 0;
}
