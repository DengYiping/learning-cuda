#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "faster_matmul.cuh"
#include <hip/hip_cooperative_groups.h>
#include "ptx.cuh"
#include <hip/hip_runtime.h> // Include CUDA Driver API header
#include <stdio.h> // For printf debugging
#include <cstdlib> // For exit
#include <algorithm> // For std::max

#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))
namespace cg = cooperative_groups;

// Helper macro for CUDA Driver API error checking
#define CHECK_CUDA_DRIVER(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        const char *err_str; \
        hipDrvGetErrorString(err, &err_str); \
        fprintf(stderr, "CUDA Driver error in %s at line %d: %s\n", __FILE__, __LINE__, err_str); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

constexpr uint TM = 4;
constexpr uint TN = 4;
constexpr uint WMITER = 2;
constexpr uint WNITER = 2;

constexpr uint WM = 32;
constexpr uint WN = 64;

constexpr uint W_SUB_M = WM / WMITER; // 16 -> 4 threads
constexpr uint W_SUB_N = WN / WNITER; // 32 -> 8 threads 

// N is the width (K) of the tile in shared memory
template <int N>
__device__ __forceinline__ float tma_swizzle_128b_load(const float* smem_ptr, uint y, uint x) {
    // 128b swizzle with 16 byte chunks (4 floats)
    int swizzled_index = y * N + ((y % 8) ^ (x / 4)) * 4 + (x % 4);

    return smem_ptr[swizzled_index];
}

template <int BM, int BN, int BK>
__global__ __launch_bounds__(BM * BN * 32 / (WM * WN)) void tma_double_buffered_matmul(
    const __grid_constant__ CUtensorMap tensor_map_A, // Pass by value with __grid_constant__
    const __grid_constant__ CUtensorMap tensor_map_B, // Pass by value with __grid_constant__
    float* __restrict__ C,
    const int M, const int N, const int K)
{
    /* ---------------- Shared memory layout (double buffered + static mbarrier) -------------
       |  A_0 (dyn) |  A_1 (dyn) |  B_0 (dyn) |  B_1 (dyn) | + mbarrier[1] (static)
       -------------------------------------------------------------------------------------*/
    // Allocate shared memory for double-buffered tiles using dynamic shared memory
    // 16384 = 16KB, align so that we can use bitwise XOR to swap buffers
    alignas(std::max(BM * BK, BN * BK) * sizeof(float)) extern __shared__ char smem_bytes[];
    // Calculate total threads per block for warp tiling
    constexpr int THREADS_PER_BLOCK = BM * BN * 32 / (WM * WN);

    // Statically allocate the mbarrier, ensuring 16-byte alignment.
    alignas(16) __shared__ uint64_t mbarrier_storage[1];
    // Get pointer to the mbarrier.
    uint64_t* mbar = mbarrier_storage;

    // Calculate the size needed for one stage's A and B tiles
    constexpr size_t smem_tile_A_bytes = BM * BK * sizeof(float);
    constexpr size_t smem_tile_B_bytes = BK * BN * sizeof(float);
    // Offset between the two buffers for A and B respectively (in bytes)
    constexpr size_t smem_offset_A = smem_tile_A_bytes;
    constexpr size_t smem_offset_B = smem_tile_B_bytes;
    // Total bytes for one stage (used for mbarrier arrive)
    constexpr size_t smem_one_stage_bytes = smem_tile_A_bytes + smem_tile_B_bytes;

    // Base pointers for the A and B double buffers in shared memory
    // Make non-const to allow swapping via XOR
    float* smem_base_A = reinterpret_cast<float*>(&smem_bytes[0]);
    // B buffers start after both A buffers
    float* smem_base_B = reinterpret_cast<float*>(&smem_bytes[2 * smem_tile_A_bytes]);

    // blockIdx.x is the block id in the N dimension, aka the column index of the block
    // blockIdx.y is the block id in the M dimension, aka the row index of the block

    // Warp-level and lane-level indexing
    const uint warpId = threadIdx.x / 32; // Each warp has 32 threads
    const uint laneId = threadIdx.x % 32; // Lane index within the warp (0-31)

    // Calculate 2D warp indices within the block
    constexpr uint warps_per_block_N = BN / WN; // Warps along the N dimension of the block tile
    const uint warp_col = warpId % warps_per_block_N; // Warp's column index within the block
    const uint warp_row = warpId / warps_per_block_N; // Warp's row index within the block

    // Calculate 2D lane indices within the warp tile (using fixed TM=4, TN=4)
    const uint lane_col = laneId % (W_SUB_N / TN); // Lane's col index within the WN tile (0-15)
    const uint lane_row = laneId / (W_SUB_N / TN); // Lane's row index within the WN tile (0-1) - check calculation

    // Each warp computes a WM x WN tile
    C += (blockIdx.y * BM + warp_row * WM) * N + (blockIdx.x * BN + warp_col * WN);

    alignas(16) float warp_results[WMITER * TM][WNITER * TN] = {0.0f};
    // Double buffer registers for M and N strips (size matches one MMA iteration)
    alignas(16) float reg_M[2][WMITER * TM]; // Holds WMITER * TM elements
    alignas(16) float reg_N[2][WNITER * TN]; // Holds WNITER * TN elements

    // ------------------------- PTX Asynchronous copy setup -------------------------
    auto block = cg::this_thread_block();
    const bool is_master_thread = (block.thread_rank() == 0);

    // Initialize the mbarrier from thread 0.
    if (is_master_thread) {
        ptx::mbarrier_init(mbar, THREADS_PER_BLOCK);
        // Fence to ensure initialization is visible to async copy units.
        ptx::fence_mbarrier_init_release_cluster();
    }
    // Sync all threads to ensure barrier is initialized before use.
    block.sync();

    // Phase variable for mbarrier wait cycles. Also used for read stage index.
    uint32_t phase = 0;

    // Register stage indices for double buffering
    int reg_read_stage = 0;
    int reg_write_stage = 0;// will be toggled before first use

    // --- Prime the register buffer for dot_idx = 0 ---
    reg_write_stage ^= 1; // Now reg_write_stage = 1

    // Calculate total bytes per tile for A and B
    constexpr size_t total_bytes_per_stage = smem_one_stage_bytes; // Reuse calculation

    // Calculate base 2D offsets for this block
    const uint32_t base_offset_A_y = blockIdx.y * BM;
    const uint32_t base_offset_B_x = blockIdx.x * BN;

    // Offset for the next tile
    uint32_t next_offset_K = 0;

    // ---------------- Prime the pipeline : load the very first tile using TMA ----------------
    if (is_master_thread) {
        // Initiate A tile copy (TMA 2D) into the initial buffer (buffer 0)
        ptx::cp_async_bulk_tensor_2d_global_to_shared(
            reinterpret_cast<uint64_t*>(smem_base_A), // Use current base pointer
            (const uint64_t*)&tensor_map_A,
            next_offset_K, base_offset_A_y,
            mbar
        );
        // Initiate B tile copy (TMA 2D) into the initial buffer (buffer 0)
        ptx::cp_async_bulk_tensor_2d_global_to_shared(
            reinterpret_cast<uint64_t*>(smem_base_B), // Use current base pointer
            (const uint64_t*)&tensor_map_B,
            base_offset_B_x, next_offset_K,
            mbar
        );
        // Master thread arrives, indicating total expected bytes for this phase.
        ptx::mbarrier_arrive_expect_tx(mbar, total_bytes_per_stage);
    } else {
        // Other threads just arrive.
        ptx::mbarrier_arrive(mbar);
    }
    // Swap pointers to point to the other buffer (buffer 1) for the next copy
    smem_base_A = reinterpret_cast<float*>(reinterpret_cast<uintptr_t>(smem_base_A) ^ smem_offset_A);
    smem_base_B = reinterpret_cast<float*>(reinterpret_cast<uintptr_t>(smem_base_B) ^ smem_offset_B);

    // number of K-tiles we will iterate over
    const uint num_tiles = CEIL_DIV(K, BK);

    for (uint tile = 0; tile < num_tiles; ++tile) {
        next_offset_K += BK;
        // Wait for the copy initiated in the previous iteration (or priming phase) to complete.
        // This copy targeted the buffer *not* currently pointed to by smem_base_A/B.
        ptx::mbarrier_wait_parity(mbar, phase);

        // ---------------- Preload the next tile (if any) while computation continues ----------------
        if (tile + 1 < num_tiles) {
            if (is_master_thread) {
                 // Initiate A tile copy (TMA 2D) for the next tile into the buffer pointed to by current smem_base_A
                ptx::cp_async_bulk_tensor_2d_global_to_shared(
                    reinterpret_cast<uint64_t*>(smem_base_A), // Use current base pointer
                    (const uint64_t*)&tensor_map_A,
                    next_offset_K, base_offset_A_y,
                    mbar
                );
                // Initiate B tile copy (TMA 2D) for the next tile into the buffer pointed to by current smem_base_B
                 ptx::cp_async_bulk_tensor_2d_global_to_shared(
                    reinterpret_cast<uint64_t*>(smem_base_B), // Use current base pointer
                    (const uint64_t*)&tensor_map_B,
                    base_offset_B_x, next_offset_K,
                    mbar
                );
                // Master thread arrives, indicating total expected bytes for the next phase.
                ptx::mbarrier_arrive_expect_tx(mbar, total_bytes_per_stage);
                if (tile + 2 < num_tiles) {
                    ptx::prefetch_async_bulk_tensor_2d_global_l2(
                        (const uint64_t*)&tensor_map_A,
                        next_offset_K + BK,
                        base_offset_A_y
                    );
                    ptx::prefetch_async_bulk_tensor_2d_global_l2(
                        (const uint64_t*)&tensor_map_B,
                        base_offset_B_x,
                        next_offset_K + BK
                    );
                }
            } else {
                // Other threads just arrive for the next phase's copy.
                ptx::mbarrier_arrive(mbar);
            }
            // Note: Global A/B pointers are not advanced here as TensorMap handles addressing
        }

        // Swap pointers to point to the other buffer (buffer 1) for the next copy
        smem_base_A = reinterpret_cast<float*>(reinterpret_cast<uintptr_t>(smem_base_A) ^ smem_offset_A);
        smem_base_B = reinterpret_cast<float*>(reinterpret_cast<uintptr_t>(smem_base_B) ^ smem_offset_B);
        // ---------------- Matrix multiply on the current shared-memory tile ----------------

        // Prime registers for dot_idx = 0 using the data from the just-completed TMA copy
        // Load A strip for dot_idx = 0 into the write stage register buffer
        #pragma unroll
        for (uint w_sub_row = 0; w_sub_row < WMITER; ++w_sub_row) {
            #pragma unroll
            for (uint i = 0; i < TM; ++i) {
                reg_M[reg_write_stage][w_sub_row * TM + i] = 
                    smem_base_A[(warp_row * WM + w_sub_row * W_SUB_M + lane_row * TM + i) * BK + 0]; // dot_idx = 0
                    // tma_swizzle_128b_load<BK>(smem_base_A, warp_row * WM + w_sub_row * W_SUB_M + lane_row * TM + i, 0); // dot_idx = 0
            }
        }
        // Load B strip for dot_idx = 0 into the write stage register buffer
        #pragma unroll
        for (uint w_sub_col = 0; w_sub_col < WNITER; ++w_sub_col) {
           #pragma unroll
           for (uint j = 0; j < TN; j += 4) { // Vectorized load assumes TN is multiple of 4
               reinterpret_cast<float4*>(&reg_N[reg_write_stage][w_sub_col * TN + j])[0] =
                   reinterpret_cast<const float4*>(&smem_base_B[0 * BN + (warp_col * WN + w_sub_col * W_SUB_N + lane_col * TN + j)])[0]; // dot_idx = 0
           }
        }


        // Iterate over the K dimension one element at a time, using two register buffers
        #pragma unroll // Might be too aggressive? Keep for now.
        for (uint dot_idx = 0; dot_idx < BK; ++dot_idx) {
            // Swap read & write buffers for the next iteration
            reg_read_stage ^= 1;
            reg_write_stage ^= 1;

            // Pre-load data that will be needed in the *next* iteration into the write buffer
            const uint next_dot_idx = dot_idx + 1;
            if (next_dot_idx < BK) {
                // Load next A strip into write buffer
                #pragma unroll
                for (uint w_sub_row = 0; w_sub_row < WMITER; ++w_sub_row) {
                    #pragma unroll
                    for (uint i = 0; i < TM; ++i) {
                        reg_M[reg_write_stage][w_sub_row * TM + i] =
                            smem_base_A[(warp_row * WM + w_sub_row * W_SUB_M + lane_row * TM + i) * BK + next_dot_idx];
                            // tma_swizzle_128b_load<BK>(smem_base_A, warp_row * WM + w_sub_row * W_SUB_M + lane_row * TM + i, next_dot_idx);
                    }
                }

                // Load next B strip into write buffer (vectorised)
                #pragma unroll
                for (uint w_sub_col = 0; w_sub_col < WNITER; ++w_sub_col) {
                    #pragma unroll
                    for (uint j = 0; j < TN; j += 4) { // Vectorised load assumes TN divisible by 4
                        reinterpret_cast<float4*>(&reg_N[reg_write_stage][w_sub_col * TN + j])[0] =
                            reinterpret_cast<const float4*>(&smem_base_B[next_dot_idx * BN +
                            (warp_col * WN + w_sub_col * W_SUB_N + lane_col * TN + j)])[0];
                    }
                }
            }
            // Compute using data that is currently in the read buffer
            #pragma unroll
            for (uint w_sub_row = 0; w_sub_row < WMITER; ++w_sub_row) {
                #pragma unroll
                for (uint w_sub_col = 0; w_sub_col < WNITER; ++w_sub_col) {
                    #pragma unroll
                    for (uint i = 0; i < TM; ++i) { // Inner loop over TM
                        #pragma unroll
                        for (uint j = 0; j < TN; ++j) { // Inner loop over TN
                            warp_results[w_sub_row * TM + i][w_sub_col * TN + j] +=
                                reg_M[reg_read_stage][w_sub_row * TM + i] *
                                reg_N[reg_read_stage][w_sub_col * TN + j];
                        }
                    }
                }
            }
        }

        // Sync threads before next iteration to ensure computation is finished before
        // potentially overwriting the shared memory buffer in the next copy phase.
        // Also ensures all threads have updated phase/write_stage.
        block.sync();

        // Toggle read_stage to the buffer we just finished copying (and will read next)
        // write_stage ^= 1; // Removed - shared memory stage is implicit in pointer swaps
        // Flip the phase for the next mbarrier wait.
        phase ^= 1;
    }

    // At this point, all tiles have been processed and results are in warp_results.
    // ---------------- Store the results to global memory ----------------
    #pragma unroll
    for (uint w_sub_row = 0; w_sub_row < WMITER; ++w_sub_row) {
        #pragma unroll
        for (uint i = 0; i < TM; ++i) { // Inner loop over TM
            #pragma unroll
            for (uint w_sub_col = 0; w_sub_col < WNITER; ++w_sub_col) {
                #pragma unroll
                for (uint j = 0; j < TN; j += 4) { // Inner loop over TN (vectorized store)
                    uint write_idx_base = (w_sub_row * W_SUB_M + lane_row * TM + i) * N + (w_sub_col * W_SUB_N + lane_col * TN + j);
                    reinterpret_cast<float4*>(&C[write_idx_base])[0] = reinterpret_cast<float4*>(&warp_results[w_sub_row * TM + i][w_sub_col * TN + j])[0];
                }
            }
        }
    }
}

// Kernel launcher function
void launch_tma_double_buffered_matmul(const float* __restrict__ d_A, const float* __restrict__ d_B, float* __restrict__ d_C, int m, int n, int k, hipStream_t stream) {
    // Use the best parameters found
    constexpr int BM = 64;
    constexpr int BN = 128;
    constexpr int BK = 32;

    // Create Tensor Maps
    CUtensorMap tensor_map_A;
    CUtensorMap tensor_map_B;

    const cuuint32_t elementStrides[] = {1, 1}; // Contiguous access

    // Tensor A (M x K) -> {inner (K), outer (M)}
    const uint64_t globalDimA[] = {(uint64_t)k, (uint64_t)m};
    // Global stride for A: stride between rows = K * sizeof(float)
    const uint64_t globalStrideA[] = {sizeof(float), (uint64_t)k * sizeof(float)};
    const cuuint32_t boxDimA[] = {BK, BM}; // {inner dim size, outer dim size}

    CHECK_CUDA_DRIVER(cuTensorMapEncodeTiled(
        &tensor_map_A,
        CU_TENSOR_MAP_DATA_TYPE_FLOAT32,
        2,                          // rank
        (void*)d_A,                 // globalAddress
        globalDimA,                 // globalDim
        globalStrideA + 1,          // globalStride (expects outer stride only)
        boxDimA,                    // boxDim
        elementStrides,             // elementStride
        CU_TENSOR_MAP_INTERLEAVE_NONE,
        // CU_TENSOR_MAP_SWIZZLE_128B, // Use 128b swizzle, 16B chunks
        CU_TENSOR_MAP_SWIZZLE_NONE,
        CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    ));

    // Tensor B (K x N) -> {inner (N), outer (K)}
    const uint64_t globalDimB[] = {(uint64_t)n, (uint64_t)k};
    // Global stride for B: stride between rows = N * sizeof(float)
    const uint64_t globalStrideB[] = {sizeof(float), (uint64_t)n * sizeof(float)};
    const cuuint32_t boxDimB[] = {BN, BK}; // {inner dim size, outer dim size}

     CHECK_CUDA_DRIVER(cuTensorMapEncodeTiled(
        &tensor_map_B,
        CU_TENSOR_MAP_DATA_TYPE_FLOAT32,
        2,                          // rank
        (void*)d_B,                 // globalAddress
        globalDimB,                 // globalDim
        globalStrideB + 1,          // globalStride (expects outer stride only)
        boxDimB,                    // boxDim
        elementStrides,             // elementStride
        CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE,
        CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    ));


    // Calculate block dimension based on warps per block
    dim3 blockDim(BM * BN * 32 / (WM * WN)); // THREADS_PER_BLOCK
    // Reversing order to optimize L2 cache access. Grid will move on the N dimension fast and M dimension slow.
    // With row-major layout, this is more cache-friendly.
    dim3 gridDim(CEIL_DIV(n, BN), CEIL_DIV(m, BM));

    // Shared memory: Calculate size needed for the double-buffered A/B tiles only.
    // Layout: A0 | A1 | B0 | B1
    size_t smem_bytes_dynamic = (2 * BM * BK + 2 * BN * BK) * sizeof(float);

    // Launch kernel with dynamic shared memory size for A/B buffers
    tma_double_buffered_matmul<BM, BN, BK><<<gridDim, blockDim, smem_bytes_dynamic, stream>>>(
        tensor_map_A, tensor_map_B, d_C, m, n, k);
}

int main() {
    // Initialize CUDA Driver API
    CHECK_CUDA_DRIVER(hipInit(0));
    hipCtx_t ctx;
    CHECK_CUDA_DRIVER(hipCtxGetCurrent(&ctx)); // Ensure context exists

    // Use the best parameters found
    constexpr int BM = 64;
    constexpr int BN = 128;
    constexpr int BK = 32;

    hipDeviceProp_t deviceProp;
    CHECK_CUDA(hipGetDeviceProperties(&deviceProp, 0));
    std::cout << "Device name: " << deviceProp.name << std::endl;
    std::cout << "Shared memory size: " << deviceProp.sharedMemPerMultiprocessor << std::endl;
    std::cout << "Shared memory size per block: " << deviceProp.sharedMemPerBlock << std::endl;

    // Check if device supports cooperative launch
    int cooperativeLaunch;
    CHECK_CUDA(hipDeviceGetAttribute(&cooperativeLaunch, hipDeviceAttributeCooperativeLaunch, 0));
    if (!cooperativeLaunch) {
        std::cerr << "Device does not support cooperative launch, which is required for TMA mbarrier." << std::endl;
        //return 1; // Or handle gracefully
    }
     // Check if device supports cluster launch needed for mbarrier
    int clusterLaunch;
    CHECK_CUDA(hipDeviceGetAttribute(&clusterLaunch, cudaDevAttrClusterLaunch, 0));
    if (!clusterLaunch) {
        std::cerr << "Device does not support cluster launch, which is required for TMA mbarrier." << std::endl;
        // return 1; // Or handle gracefully
    }
     // Check if device supports asynchronous copy specifically
    int asyncEngineCount;
    CHECK_CUDA(hipDeviceGetAttribute(&asyncEngineCount, hipDeviceAttributeAsyncEngineCount, 0));
    if (asyncEngineCount == 0) {
        std::cerr << "Device does not have async copy engines required for TMA." << std::endl;
       // return 1; // Or handle gracefully
    }

     std::cout << "Device supports cooperative launch: " << (cooperativeLaunch ? "Yes" : "No") << std::endl;
     std::cout << "Device supports cluster launch: " << (clusterLaunch ? "Yes" : "No") << std::endl;
     std::cout << "Device async engine count: " << asyncEngineCount << std::endl;


    // Calculate required dynamic shared memory for A/B buffers
    size_t required_smem_dynamic = (2 * BM * BK + 2 * BN * BK) * sizeof(float); // Updated calculation
    std::cout << "Required dynamic shared memory per block: " << required_smem_dynamic << " bytes" << std::endl;
    // Calculate total static shared memory (mbarrier only)
    size_t required_smem_static = sizeof(uint64_t); // Size of the static mbarrier
    std::cout << "Required static shared memory per block: " << required_smem_static << " bytes" << std::endl;

    // Check if required dynamic shared memory exceeds limit
    if (required_smem_dynamic + required_smem_static > deviceProp.sharedMemPerMultiprocessor) {
         std::cerr << "Error: Required dynamic shared memory (" << required_smem_dynamic
                   << " bytes) plus static shared memory (" << required_smem_static
                   << " bytes) exceeds device limit per multiprocessor (" << deviceProp.sharedMemPerMultiprocessor
                   << " bytes)." << std::endl;
        return 1;
    }

    // Set shared memory carveout for this kernel - potentially higher if needed for TMA
    CHECK_CUDA(hipFuncSetAttribute(
        (const void*)tma_double_buffered_matmul<BM, BN, BK>,
        hipFuncAttributePreferredSharedMemoryCarveout,
        100 // Max carveout, as TMA benefits from L1
    ));
    // Set dynamic shared memory size attribute for the kernel
    CHECK_CUDA(hipFuncSetAttribute(
        (const void*)tma_double_buffered_matmul<BM, BN, BK>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        static_cast<int>(required_smem_dynamic)
    ));

    // Default matrix dimensions
    int m = 4096; // Matrix A: m x k
    int n = 2048; // Matrix B: k x n, Matrix C: m x n
    int k = 512;

    std::cout << "Running TMA Double Buffered matrix multiplication benchmark:" << std::endl;

    // Run the benchmark with the new kernel
    float avg_time = run_benchmark<float>(
        launch_tma_double_buffered_matmul, m, n, k
    );

    return 0;
} 